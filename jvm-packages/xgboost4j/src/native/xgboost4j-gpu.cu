/**
 * Copyright 2021-2024, XGBoost Contributors
 */
#include <jni.h>
#include <xgboost/c_api.h>

#include "../../../../src/common/cuda_pinned_allocator.h"
#include "../../../../src/common/device_vector.cuh"  // for device_vector
#include "../../../../src/data/array_interface.h"
#include "../../../../src/c_api/c_api_error.h"
#include "jvm_utils.h"
#include <fstream>


namespace xgboost {
namespace jni {

template <typename T, typename Alloc>
T const *RawPtr(std::vector<T, Alloc> const &data) {
  return data.data();
}

template <typename T, typename Alloc> T *RawPtr(std::vector<T, Alloc> &data) {
  return data.data();
}

template <typename T> T const *RawPtr(dh::device_vector<T> const &data) {
  return data.data().get();
}

template <typename T> T *RawPtr(dh::device_vector<T> &data) {
  return data.data().get();
}

template <typename T> T CheckJvmCall(T const &v, JNIEnv *jenv) {
  if (!v) {
    CHECK(jenv->ExceptionOccurred());
    jenv->ExceptionDescribe();
  }
  return v;
}

template <typename VCont>
void CopyColumnMask(xgboost::ArrayInterface<1> const &interface,
                    std::vector<Json> const &columns, hipMemcpyKind kind,
                    size_t c, VCont *p_mask, Json *p_out, hipStream_t stream) {
  auto &mask = *p_mask;
  auto &out = *p_out;
  auto size = sizeof(typename VCont::value_type) * interface.n;
  mask.resize(size);
  CHECK(RawPtr(mask));
  CHECK(size);
  CHECK(interface.valid.Data());
  dh::safe_cuda(
      hipMemcpyAsync(RawPtr(mask), interface.valid.Data(), size, kind, stream));
  auto const &mask_column = columns[c]["mask"];
  out["mask"] = Object();
  std::vector<Json> mask_data{
      Json{reinterpret_cast<Integer::Int>(RawPtr(mask))},
      Json{get<Boolean const>(mask_column["data"][1])}};
  out["mask"]["data"] = Array(std::move(mask_data));
  if (get<Array const>(mask_column["shape"]).size() == 2) {
    std::vector<Json> mask_shape{
        Json{get<Integer const>(mask_column["shape"][0])},
        Json{get<Integer const>(mask_column["shape"][1])}};
    out["mask"]["shape"] = Array(std::move(mask_shape));
  } else if (get<Array const>(mask_column["shape"]).size() == 1) {
    std::vector<Json> mask_shape{
        Json{get<Integer const>(mask_column["shape"][0])}};
    out["mask"]["shape"] = Array(std::move(mask_shape));
  } else {
    LOG(FATAL) << "Invalid shape of mask";
  }
  out["mask"]["typestr"] = String("<t1");
  out["mask"]["version"] = Integer(3);
}

template <typename DCont, typename VCont>
void CopyInterface(std::vector<xgboost::ArrayInterface<1>> &interface_arr,
                   std::vector<Json> const &columns, hipMemcpyKind kind,
                   std::vector<DCont> *p_data, std::vector<VCont> *p_mask,
                   std::vector<xgboost::Json> *p_out, hipStream_t stream) {
  p_data->resize(interface_arr.size());
  p_mask->resize(interface_arr.size());
  p_out->resize(interface_arr.size());
  for (size_t c = 0; c < interface_arr.size(); ++c) {
    auto &interface = interface_arr.at(c);
    size_t element_size = interface.ElementSize();
    size_t size = element_size * interface.n;

    auto &data = (*p_data)[c];
    auto &mask = (*p_mask)[c];
    data.resize(size);
    dh::safe_cuda(hipMemcpyAsync(RawPtr(data), interface.data, size, kind, stream));

    auto &out = (*p_out)[c];
    out = Object();
    std::vector<Json> j_data{
        Json{Integer(reinterpret_cast<Integer::Int>(RawPtr(data)))},
        Json{Boolean{false}}};

    out["data"] = Array(std::move(j_data));
    out["shape"] = Array(std::vector<Json>{Json(Integer(interface.Shape<0>()))});

    if (interface.valid.Data()) {
      CopyColumnMask(interface, columns, kind, c, &mask, &out, stream);
    }
    out["typestr"] = String("<f4");
    out["version"] = Integer(3);
  }
}

template <typename T>
void CopyMetaInfo(Json *p_interface, dh::device_vector<T> *out, hipStream_t stream) {
  auto &j_interface = *p_interface;
  CHECK_EQ(get<Array const>(j_interface).size(), 1);
  auto object = get<Object>(get<Array>(j_interface)[0]);
  ArrayInterface<1> interface(object);
  out->resize(interface.Shape<0>());
  size_t element_size = interface.ElementSize();
  size_t size = element_size * interface.n;
  dh::safe_cuda(hipMemcpyAsync(RawPtr(*out), interface.data, size,
                                hipMemcpyDeviceToDevice, stream));
  j_interface[0]["data"][0] = reinterpret_cast<Integer::Int>(RawPtr(*out));
}

template <typename DCont, typename VCont> struct DataFrame {
  std::vector<DCont> data;
  std::vector<VCont> valid;
  std::vector<Json> interfaces;
};

// The base class for external memory
class ExternalMemory {
 public:
  // Load data from the exact external memory to the GPU
  virtual void LoadData(size_t batch_number,
                        std::vector<xgboost::Json> *p_out,
                        hipStream_t stream) = 0;

  // Stage data into the exact external memory from GPU.
  virtual void StageData(
      std::vector<xgboost::ArrayInterface<1>> &interface_arr,
      std::vector<Json> const &columns,
      hipStream_t stream) = 0;

  virtual ~ExternalMemory() = default;

 protected:
  // Temp buffer on device, each `dh::device_vector` represents a column from cudf.
  std::vector<dh::device_vector<char>> staging_data_;
  std::vector<dh::device_vector<uint8_t>> staging_mask_;
};

// The data will be stored on CPU memory
class HostExternalMemory: public ExternalMemory {
 public:
  void StageData(std::vector<xgboost::ArrayInterface<1>> &interfaces,
                 std::vector<Json> const &columns,
                 hipStream_t stream) override {
    std::cerr << "HostExternalMemory StageData batch number: " << std::endl;
    // DataFrame
    using T = decltype(host_columns_)::value_type::element_type;
    host_columns_.emplace_back(std::unique_ptr<T>(new T));

    CopyInterface(interfaces, columns, hipMemcpyDeviceToHost, &host_columns_.back()->data,
                  &host_columns_.back()->valid, &host_columns_.back()->interfaces, stream);
  }

  void LoadData(size_t batch_number,
                std::vector<xgboost::Json> *p_out,
                hipStream_t stream) override {
    std::cerr << "HostExternalMemory LoadData batch number: " << batch_number << std::endl;
    // Data
    auto const &json_interface = host_columns_.at(batch_number)->interfaces;
    std::vector<ArrayInterface<1>> in;
    for (auto interface : json_interface) {
      auto column = ArrayInterface<1>(get<Object const>(interface));
      in.emplace_back(column);
    }
    CopyInterface(in, json_interface, hipMemcpyHostToDevice, &staging_data_,
                  &staging_mask_, p_out, nullptr);
  }

 private:
    template <typename T>
    using Alloc = xgboost::common::cuda_impl::PinnedAllocator<T>;
    template <typename U>
    using HostVector = std::vector<U, Alloc<U>>;
    // This vector is created for staging device data on host to save GPU memory.
    // When space is not of concern, we can stage them on device memory directly.
    std::vector<
        std::unique_ptr<DataFrame<HostVector<char>, HostVector<std::uint8_t>>>>
        host_columns_;
};

struct DataFile {
  std::string path;
  size_t size;
  size_t shape;
};

struct FilesDataFrame {
  std::vector<DataFile> data_files;
  std::vector<DataFile> valid_files;
};

// The data will be cached into local disk.
class DiskExternalMemory: public ExternalMemory {
 public:
  DiskExternalMemory(std::string root): root_(root) {}

  void LoadData(size_t batch_number, std::vector<xgboost::Json> *p_out,
                hipStream_t stream) override {
    std::cerr << " DiskExternalMemory LoadData " << std::endl;
    auto files_data_frame = staged_files_.at(batch_number);
    auto col_number = files_data_frame.data_files.size();

    p_out->resize(col_number);
    staging_data_.resize(col_number);

    for (size_t c = 0; c < col_number; c++) {
      auto data_file = files_data_frame.data_files.at(c);

      std::vector<char> host_data;
      host_data.resize(data_file.size);
      std::ifstream inputFile(data_file.path, std::ios::binary);
      if (inputFile.is_open()) {
        inputFile.read(RawPtr(host_data), data_file.size);
        inputFile.close();
      } else {
        std::cerr << "Failed to open file for reading" << std::endl;
      }

      auto device_data = staging_data_.at(c);
      device_data.resize(data_file.size);
      dh::safe_cuda(hipMemcpyAsync(RawPtr(device_data), RawPtr(host_data), data_file.size,
                                    hipMemcpyHostToDevice, stream));
      auto &out = (*p_out).at(c);
      out = Object();
      std::vector<Json> j_data{Json{Integer(reinterpret_cast<Integer::Int>(RawPtr(device_data)))},
                               Json{Boolean{false}}};

      out["data"] = Array(std::move(j_data));
      out["shape"] = Array(std::vector<Json>{Json(Integer(data_file.shape))});
      out["typestr"] = String("<f4");
      out["version"] = Integer(3);
      // TODO support mask
    }
  }

  void StageData(std::vector<xgboost::ArrayInterface<1>> &interface_arr,
                 const std::vector<Json> &columns, hipStream_t stream) override {
    std::cerr << " DiskExternalMemory StageData " << std::endl;
    ++n_batches_;
    FilesDataFrame files_data_frame;
    for (size_t c = 0; c < interface_arr.size(); ++c) {
      auto &interface = interface_arr.at(c);
      size_t element_size = interface.ElementSize();
      size_t size = element_size * interface.n;

      std::vector<char> tmp;
      tmp.resize(size);
      dh::safe_cuda(hipMemcpyAsync(RawPtr(tmp), interface.data, size,
                                    hipMemcpyDeviceToHost, stream));

      std::string file_name = GenerateFileName(n_batches_, c, "data");
      WriteDataToFile(file_name, RawPtr(tmp), size);

      // Store the information.
      files_data_frame.data_files.emplace_back(DataFile{file_name, size, interface.Shape<0>()});

      if (interface.valid.Data()) {
        // TODO support mask
      }
    }
    staged_files_.emplace_back(files_data_frame);
  }

  ~DiskExternalMemory() override {
    for (auto cached_files: staged_files_) {
        for (auto data_file : cached_files.data_files) {
            std::cerr << "removing " << data_file.path.c_str() << std::endl;
            std::remove(data_file.path.c_str());
        }
        for (auto data_file : cached_files.valid_files) {
            std::remove(data_file.path.c_str());
        }
    }
  };

 private:
  std::string GenerateFileName(size_t batch_number, size_t column, std::string type) const {
      std::stringstream ss;
      ss << root_ << "/" << batch_number << "_" << column << "_" << type << ".bin";
      return ss.str();
  }

  void WriteDataToFile(std::string path, char *data, size_t size) const {
    std::ofstream output_file(path, std::ios::binary);
    if (output_file.is_open()) {
      output_file.write(data, size);
      output_file.close();
    } else {
      std::cerr << "Failed to open file " << std::endl;
    }
  }

  std::vector<FilesDataFrame> staged_files_;
  const std::string root_; // the root path of external memory
  size_t n_batches_ = 0;
};

class DataIteratorProxy {
  DMatrixHandle proxy_;
  JNIEnv *jenv_;
  int jni_status_;
  jobject jiter_;
  std::unique_ptr<ExternalMemory> ext_memory_;

  // Staging area for metainfo.
  // TODO(Bobby): label_upper_bound, label_lower_bound, group.
  std::vector<std::unique_ptr<dh::device_vector<float>>> labels_;
  std::vector<std::unique_ptr<dh::device_vector<float>>> weights_;
  std::vector<std::unique_ptr<dh::device_vector<float>>> base_margins_;
  std::vector<std::unique_ptr<dh::device_vector<int>>> qids_;
  std::vector<Json> label_interfaces_;
  std::vector<Json> weight_interfaces_;
  std::vector<Json> margin_interfaces_;
  std::vector<Json> qid_interfaces_;

  size_t it_{0};
  size_t n_batches_{0};
  bool initialized_{false};
  jobject last_batch_ {nullptr};

  hipStream_t copy_stream_;

 public:
  explicit DataIteratorProxy(jobject jiter, std::string external_path): jiter_{jiter} {
    if (!external_path.empty()) {
      ext_memory_ = std::make_unique<DiskExternalMemory>(external_path);
    } else {
      ext_memory_ = std::make_unique<HostExternalMemory>();
    }
    XGProxyDMatrixCreate(&proxy_);
    jni_status_ =
        GlobalJvm()->GetEnv(reinterpret_cast<void **>(&jenv_), JNI_VERSION_1_6);
    this->Reset();
    dh::safe_cuda(hipStreamCreateWithFlags(&copy_stream_, hipStreamNonBlocking));
  }
  ~DataIteratorProxy() {
    XGDMatrixFree(proxy_);
    dh::safe_cuda(hipStreamDestroy(copy_stream_));
  }

  DMatrixHandle GetDMatrixHandle() const { return proxy_; }

  // Helper function for staging meta info.
  void StageMetaInfo(Json json_interface) {
    CHECK(!IsA<Null>(json_interface));
    auto json_map = get<Object const>(json_interface);
    if (json_map.find("label") == json_map.cend()) {
      LOG(FATAL) << "Must have a label field.";
    }

    Json label = json_interface["label"];
    CHECK(!IsA<Null>(label));
    labels_.emplace_back(new dh::device_vector<float>);
    CopyMetaInfo(&label, labels_.back().get(), copy_stream_);
    label_interfaces_.emplace_back(label);

    std::string str;
    Json::Dump(label, &str);
    XGDMatrixSetInfoFromInterface(proxy_, "label", str.c_str());

    if (json_map.find("weight") != json_map.cend()) {
      Json weight = json_interface["weight"];
      CHECK(!IsA<Null>(weight));
      weights_.emplace_back(new dh::device_vector<float>);
      CopyMetaInfo(&weight, weights_.back().get(), copy_stream_);
      weight_interfaces_.emplace_back(weight);

      Json::Dump(weight, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "weight", str.c_str());
    }

    if (json_map.find("baseMargin") != json_map.cend()) {
      Json basemargin = json_interface["baseMargin"];
      base_margins_.emplace_back(new dh::device_vector<float>);
      CopyMetaInfo(&basemargin, base_margins_.back().get(), copy_stream_);
      margin_interfaces_.emplace_back(basemargin);

      Json::Dump(basemargin, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "base_margin", str.c_str());
    }

    if (json_map.find("qid") != json_map.cend()) {
      Json qid = json_interface["qid"];
      qids_.emplace_back(new dh::device_vector<int>);
      CopyMetaInfo(&qid, qids_.back().get(), copy_stream_);
      qid_interfaces_.emplace_back(qid);

      Json::Dump(qid, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "qid", str.c_str());
    }
  }

  void CloseJvmBatch() {
    if (last_batch_) {
      jclass batch_class = CheckJvmCall(jenv_->GetObjectClass(last_batch_), jenv_);
      jmethodID closeMethod = CheckJvmCall(jenv_->GetMethodID(batch_class, "close", "()V"), jenv_);
      jenv_->CallVoidMethod(last_batch_, closeMethod);
      last_batch_ = nullptr;
    }
  }

  void Reset() {
    it_ = 0;
    this->CloseJvmBatch();
  }

  int32_t PullIterFromJVM() {
    jclass iterClass = jenv_->FindClass("java/util/Iterator");
    this->CloseJvmBatch();

    jmethodID has_next =
        CheckJvmCall(jenv_->GetMethodID(iterClass, "hasNext", "()Z"), jenv_);
    jmethodID next = CheckJvmCall(
        jenv_->GetMethodID(iterClass, "next", "()Ljava/lang/Object;"), jenv_);

    if (jenv_->CallBooleanMethod(jiter_, has_next)) {
      // batch should be ColumnBatch from jvm
      jobject batch = CheckJvmCall(jenv_->CallObjectMethod(jiter_, next), jenv_);
      jclass batch_class = CheckJvmCall(jenv_->GetObjectClass(batch), jenv_);
      jmethodID toJson = CheckJvmCall(jenv_->GetMethodID(
        batch_class, "toJson", "()Ljava/lang/String;"), jenv_);

      auto jinterface =
        static_cast<jstring>(jenv_->CallObjectMethod(batch, toJson));
      CheckJvmCall(jinterface, jenv_);
      char const *c_interface_str =
          CheckJvmCall(jenv_->GetStringUTFChars(jinterface, nullptr), jenv_);

      StageData(c_interface_str);

      jenv_->ReleaseStringUTFChars(jinterface, c_interface_str);

      last_batch_ = batch;
      return 1;
    } else {
      return 0;
    }
  }

  void StageData(std::string interface_str) {
    ++n_batches_;

    // Stage the meta info.
    auto json_interface =
        Json::Load({interface_str.c_str(), interface_str.size()});
    CHECK(!IsA<Null>(json_interface));
    StageMetaInfo(json_interface);

    Json features = json_interface["features"];
    auto json_columns = get<Array const>(features);
    std::vector<ArrayInterface<1>> interfaces;

    // Stage the data
    for (auto &json_col : json_columns) {
      auto column = ArrayInterface<1>(get<Object const>(json_col));
      interfaces.emplace_back(column);
    }
    ext_memory_->StageData(interfaces, json_columns, copy_stream_);

    Json::Dump(features, &interface_str);
    XGProxyDMatrixSetDataCudaColumnar(proxy_, interface_str.c_str());
    it_++;
  }

  int NextFirstLoop() {
    try {
      dh::safe_cuda(hipStreamSynchronize(copy_stream_));
      if (this->PullIterFromJVM()) {
        return 1;
      } else {
        initialized_ = true;
        return 0;
      }
    } catch (dmlc::Error const &e) {
      if (jni_status_ == JNI_EDETACHED) {
        GlobalJvm()->DetachCurrentThread();
      }
      LOG(FATAL) << e.what();
    }
    LOG(FATAL) << "Unreachable";
    return 1;
  }

  int NextSecondLoop() {
    std::string str;
    // Meta
    auto const &label = this->label_interfaces_.at(it_);
    Json::Dump(label, &str);
    XGDMatrixSetInfoFromInterface(proxy_, "label", str.c_str());

    if (n_batches_ == this->weight_interfaces_.size()) {
      auto const &weight = this->weight_interfaces_.at(it_);
      Json::Dump(weight, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "weight", str.c_str());
    }

    if (n_batches_ == this->margin_interfaces_.size()) {
      auto const &base_margin = this->margin_interfaces_.at(it_);
      Json::Dump(base_margin, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "base_margin", str.c_str());
    }

    if (n_batches_ == this->qid_interfaces_.size()) {
      auto const &qid = this->qid_interfaces_.at(it_);
      Json::Dump(qid, &str);
      XGDMatrixSetInfoFromInterface(proxy_, "qid", str.c_str());
    }

    std::vector<Json> out;
    ext_memory_->LoadData(it_, &out, copy_stream_);

    Json temp{Array(std::move(out))};
    std::string interface_str;
    Json::Dump(temp, &interface_str);
    std::cerr << "NextSecondLoop " << interface_str << std::endl;
    XGProxyDMatrixSetDataCudaColumnar(proxy_, interface_str.c_str());
    it_++;
    return 1;
  }

  int Next() {
    if (!initialized_) {
      return NextFirstLoop();
    } else {
      if (it_ == n_batches_) {
        return 0;
      }
      return NextSecondLoop();
    }
  };
};

namespace {
void Reset(DataIterHandle self) {
  static_cast<xgboost::jni::DataIteratorProxy *>(self)->Reset();
}

int Next(DataIterHandle self) {
  return static_cast<xgboost::jni::DataIteratorProxy *>(self)->Next();
}

template <typename T>
using Deleter = std::function<void(T *)>;
} // anonymous namespace

XGB_DLL int XGQuantileDMatrixCreateFromCallbackImpl(JNIEnv *jenv, jclass, jobject jdata_iter,
                                                    jlongArray jref, char const *config,
                                                    jlongArray jout) {
  xgboost_CHECK_C_ARG_PTR(config);
  auto jconfig = Json::Load(StringView{config});
  auto ext_mem_path = OptionalArg<String>(jconfig, "external_memory_path", std::string(""));
  std::cerr << "XGQuantileDMatrixCreateFromCallbackImpl external_memory_path " << ext_mem_path << std::endl;

  xgboost::jni::DataIteratorProxy proxy(jdata_iter, ext_mem_path);
  DMatrixHandle result;
  DMatrixHandle ref{nullptr};

  if (jref != nullptr) {
    std::unique_ptr<jlong, Deleter<jlong>> refptr{jenv->GetLongArrayElements(jref, nullptr),
                                                  [&](jlong *ptr) {
                                                    jenv->ReleaseLongArrayElements(jref, ptr, 0);
                                                    jenv->DeleteLocalRef(jref);
                                                  }};
    ref = reinterpret_cast<DMatrixHandle>(refptr.get()[0]);
  }

  auto ret = XGQuantileDMatrixCreateFromCallback(&proxy, proxy.GetDMatrixHandle(), ref, Reset, Next,
                                                 config, &result);
  setHandle(jenv, jout, result);
  return ret;
}
} // namespace jni
} // namespace xgboost
