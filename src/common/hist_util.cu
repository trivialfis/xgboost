#include "hip/hip_runtime.h"
/*!
 * Copyright 2018~2020 XGBoost contributors
 */

#include <xgboost/logging.h>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

#include <memory>
#include <mutex>
#include <utility>
#include <vector>

#include "device_helpers.cuh"
#include "hist_util.h"
#include "hist_util.cuh"
#include "math.h"  // NOLINT
#include "quantile.h"
#include "categorical.h"
#include "xgboost/host_device_vector.h"


namespace xgboost {
namespace common {

constexpr float SketchContainer::kFactor;

namespace detail {
size_t RequiredSampleCutsPerColumn(int max_bins_per_feature, size_t num_rows) {
  double eps = 1.0 / (WQSketch::kFactor * max_bins_per_feature);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
      num_rows, eps, &dummy_nlevel, &num_cuts);
  return std::min(num_cuts, num_rows);
}

size_t RequiredSampleCuts(bst_row_t num_rows, bst_feature_t num_columns,
                          size_t max_bins_per_feature, size_t nnz) {
  auto per_column = RequiredSampleCutsPerColumn(max_bins_per_feature, num_rows);
  auto if_dense = num_columns * per_column;
  auto result = std::min(nnz, if_dense);
  return result;
}

size_t ConstantMemoryPerWindow(size_t num_rows, bst_feature_t num_columns,
                               size_t max_bins_per_feature, size_t nnz) {
  // 0. Allocate cut pointer in quantile container by increasing: n_columns + 1
  size_t total = (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 3. Allocate colomn size scan by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 4. Allocate cut pointer by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  total +=
      RequiredSampleCuts(num_rows, num_columns, max_bins_per_feature, nnz) *
      sizeof(SketchEntry);
  return total;
}

void SortByWeight(dh::device_vector<float>* weights,
                  dh::device_vector<Entry>* sorted_entries) {
  // Sort both entries and wegihts.
  dh::XGBDeviceAllocator<char> alloc;
  thrust::sort_by_key(thrust::cuda::par(alloc), sorted_entries->begin(),
                      sorted_entries->end(), weights->begin(),
                      detail::EntryCompareOp());

  // Scan weights
  dh::XGBCachingDeviceAllocator<char> caching;
  thrust::inclusive_scan_by_key(thrust::cuda::par(caching),
                                sorted_entries->begin(), sorted_entries->end(),
                                weights->begin(), weights->begin(),
                                [=] __device__(const Entry& a, const Entry& b) {
                                  return a.index == b.index;
                                });
}

struct IsCatOp {
  XGBOOST_DEVICE bool operator()(FeatureType ft) { return ft == FeatureType::kCategorical; }
};

void RemoveDuplicatedCategories(
    int32_t device, MetaInfo const &info, Span<bst_row_t> d_cuts_ptr,
    dh::device_vector<Entry> *p_sorted_entries,
    dh::caching_device_vector<size_t>* p_column_sizes_scan) {
  auto d_feature_types = info.feature_types.ConstDeviceSpan();
  auto& column_sizes_scan = *p_column_sizes_scan;
  if (!info.feature_types.Empty() &&
      thrust::any_of(dh::tbegin(d_feature_types), dh::tend(d_feature_types),
                     IsCatOp{})) {
    auto& sorted_entries = *p_sorted_entries;
    // Removing duplicated entries in categorical features.
    dh::caching_device_vector<size_t> new_column_scan(column_sizes_scan.size());
    dh::SegmentedUnique(
        column_sizes_scan.data().get(),
        column_sizes_scan.data().get() + column_sizes_scan.size(),
        sorted_entries.begin(), sorted_entries.end(),
        new_column_scan.data().get(), sorted_entries.begin(),
        [=] __device__(Entry const &l, Entry const &r) {
          if (l.index == r.index) {
            if (IsCat(d_feature_types, l.index)) {
              return l.fvalue == r.fvalue;
            }
          }
          return false;
        });

    // Renew the column scan and cut scan based on categorical data.
    auto d_old_column_sizes_scan = dh::ToSpan(column_sizes_scan);
    dh::caching_device_vector<SketchContainer::OffsetT> new_cuts_size(
        info.num_col_ + 1);
    auto d_new_cuts_size = dh::ToSpan(new_cuts_size);
    auto d_new_columns_ptr = dh::ToSpan(new_column_scan);
    CHECK_EQ(new_column_scan.size(), new_cuts_size.size());
    dh::LaunchN(device, new_column_scan.size(), [=] __device__(size_t idx) {
      d_old_column_sizes_scan[idx] = d_new_columns_ptr[idx];
      if (idx == d_new_columns_ptr.size() - 1) {
        return;
      }
      if (IsCat(d_feature_types, idx)) {
        // Cut size is the same as number of categories in input.
        d_new_cuts_size[idx] =
            d_new_columns_ptr[idx + 1] - d_new_columns_ptr[idx];
      } else {
        d_new_cuts_size[idx] = d_cuts_ptr[idx] - d_cuts_ptr[idx];
      }
    });
    // Turn size into ptr.
    thrust::exclusive_scan(thrust::device, new_cuts_size.cbegin(),
                           new_cuts_size.cend(), d_cuts_ptr.data());
  }
}

size_t EstimateBatchSize(size_t num_rows, bst_feature_t num_columns,
                         size_t max_bins_per_feature, size_t nnz,
                         size_t memory_limit, int32_t device, bool weighted) {
  // assuming dense cuts
  int64_t avail =
      memory_limit == 0 ? dh::AvailableMemory(device) : memory_limit;
  size_t lower_bound =
      ConstantMemoryPerWindow(num_rows, num_columns, max_bins_per_feature, nnz);
  CHECK_LT(static_cast<int64_t>(lower_bound), avail);
  int64_t a = static_cast<int64_t>(BytesPerElement(weighted));
  int64_t c = (-avail + static_cast<int64_t>(lower_bound));
  size_t n_elements = std::sqrt(-4 * a * c) / 2 * a;
  return n_elements;
}
}  // namespace detail

void ProcessBatch(int device, MetaInfo const &info, const SparsePage &page,
                  size_t begin, size_t end, SketchContainer *sketch_container,
                  int num_cuts_per_feature, size_t num_columns) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto& host_data = page.data.ConstHostVector();
  dh::device_vector<Entry> sorted_entries(host_data.begin() + begin,
                                          host_data.begin() + end);
  thrust::sort(thrust::cuda::par(alloc), sorted_entries.begin(),
               sorted_entries.end(), detail::EntryCompareOp());

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scanning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);
  auto d_cuts_ptr = cuts_ptr.DeviceSpan();
  detail::RemoveDuplicatedCategories(device, info, d_cuts_ptr, &sorted_entries,
                                     &column_sizes_scan);

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();
  CHECK_EQ(d_cuts_ptr.size(), column_sizes_scan.size());

  // add cuts into sketches
  sketch_container->Push(dh::ToSpan(sorted_entries), dh::ToSpan(column_sizes_scan),
                         d_cuts_ptr, h_cuts_ptr.back());
  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
  CHECK_EQ(sorted_entries.capacity(), 0);
  CHECK_NE(cuts_ptr.Size(), 0);
}

void ProcessWeightedBatch(int device, const SparsePage& page,
                          MetaInfo const& info, size_t begin, size_t end,
                          SketchContainer* sketch_container, int num_cuts_per_feature,
                          size_t num_columns,
                          bool is_ranking, Span<bst_group_t const> d_group_ptr) {
  auto weights = info.weights_.ConstDeviceSpan();

  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto& host_data = page.data.ConstHostVector();
  dh::device_vector<Entry> sorted_entries(host_data.begin() + begin,
                                          host_data.begin() + end);

  // Binary search to assign weights to each element
  dh::device_vector<float> temp_weights(sorted_entries.size());
  auto d_temp_weights = temp_weights.data().get();
  page.offset.SetDevice(device);
  auto row_ptrs = page.offset.ConstDeviceSpan();
  size_t base_rowid = page.base_rowid;
  if (is_ranking) {
    CHECK_GE(d_group_ptr.size(), 2)
        << "Must have at least 1 group for ranking.";
    CHECK_EQ(weights.size(), d_group_ptr.size() - 1)
        << "Weight size should equal to number of groups.";
    dh::LaunchN(device, temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        bst_group_t group_idx = dh::SegmentId(d_group_ptr, ridx + base_rowid);
        d_temp_weights[idx] = weights[group_idx];
      });
  } else {
    dh::LaunchN(device, temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        d_temp_weights[idx] = weights[ridx + base_rowid];
      });
  }
  detail::SortByWeight(&temp_weights, &sorted_entries);

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scaning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);
  auto d_cuts_ptr = cuts_ptr.DeviceSpan();
  detail::RemoveDuplicatedCategories(device, info, d_cuts_ptr, &sorted_entries,
                                     &column_sizes_scan);
  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();

  // Extract cuts
  sketch_container->Push(dh::ToSpan(sorted_entries),
                         dh::ToSpan(column_sizes_scan), d_cuts_ptr,
                         h_cuts_ptr.back(), dh::ToSpan(temp_weights));
  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
}

HistogramCuts DeviceSketch(int device, DMatrix* dmat, int max_bins, size_t memory_limit) {
  dmat->Info().feature_types.SetDevice(device);
  dmat->Info().feature_types.ConstDevicePointer();  // pull to device early
  // Configure batch size based on available memory
  bool has_weights = dmat->Info().weights_.Size() > 0;
  size_t num_cuts_per_feature =
      detail::RequiredSampleCutsPerColumn(max_bins, dmat->Info().num_row_);

  HistogramCuts cuts;
  SketchContainer sketch_container(dmat->Info().feature_types, max_bins,
                                   dmat->Info().num_col_, dmat->Info().num_row_,
                                   device);

  dmat->Info().weights_.SetDevice(device);
  for (const auto &batch : dmat->GetBatches<SparsePage>()) {
    size_t batch_nnz = batch.data.Size();
    auto const &info = dmat->Info();

    size_t remaining = batch_nnz;
    size_t begin = 0;

    do {
      size_t sketch_batch_num_elements = detail::EstimateBatchSize(
          info.num_row_, info.num_col_, num_cuts_per_feature, info.num_nonzero_,
          memory_limit, device, has_weights);
      size_t end =
          std::min(batch_nnz, size_t(begin + sketch_batch_num_elements));

      if (has_weights) {
        bool is_ranking = HostSketchContainer::UseGroup(dmat->Info());
        dh::caching_device_vector<uint32_t> groups(info.group_ptr_.cbegin(),
                                                   info.group_ptr_.cend());
        ProcessWeightedBatch(device, batch, dmat->Info(), begin, end,
                             &sketch_container, num_cuts_per_feature,
                             dmat->Info().num_col_, is_ranking,
                             dh::ToSpan(groups));
      } else {
        ProcessBatch(device, dmat->Info(), batch, begin, end, &sketch_container,
                     num_cuts_per_feature, dmat->Info().num_col_);
      }
      remaining -= (end - begin);
      begin = end;
    } while (remaining > 0);
  }
  sketch_container.MakeCuts(&cuts);
  return cuts;
}
}  // namespace common
}  // namespace xgboost
