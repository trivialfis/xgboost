/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include <limits>
#include "evaluate_splits.cuh"
#include "../../common/categorical.h"
#include "../../common/algorithm.cuh"

namespace xgboost {
namespace tree {
namespace {

template <typename GradientSumT>
struct ScanElem {
  size_t idx;
  GradientSumT grad;
  DeviceSplitCandidate candidate;

  ScanElem() = default;
  XGBOOST_DEVICE ScanElem(size_t _idx, GradientSumT _grad,
                          DeviceSplitCandidate _candidate)
      : idx{_idx}, grad{_grad}, candidate{_candidate} {}
  XGBOOST_DEVICE explicit ScanElem(thrust::tuple<size_t, GradientSumT, DeviceSplitCandidate> const& tu) {
    idx = thrust::get<0>(tu);
    grad = thrust::get<1>(tu);
    candidate = thrust::get<2>(tu);
  }
};

// FIXME: add bool need_backward.
template <typename GradientSumT, typename ItemTy = ScanElem<GradientSumT>>
struct ScanOp : public thrust::binary_function<ItemTy, ItemTy, ItemTy> {
  EvaluateSplitInputs<GradientSumT> left;
  EvaluateSplitInputs<GradientSumT> right;
  TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator;

  XGBOOST_DEVICE ScanOp(EvaluateSplitInputs<GradientSumT> l,
                        EvaluateSplitInputs<GradientSumT> r,
                        TreeEvaluator::SplitEvaluator<GPUTrainingParam> e)
      : left{std::move(l)}, right{std::move(r)}, evaluator{std::move(e)} {}

  template <bool forward, bool is_cat>
  XGBOOST_DEVICE DeviceSplitCandidate
  DoIt(EvaluateSplitInputs<GradientSumT> input, size_t idx,
       GradientSumT l_gpair, GradientSumT r_gpair,
       DeviceSplitCandidate l_split, bst_feature_t fidx) const {
    DeviceSplitCandidate best;
    float gain = evaluator.CalcSplitGain(
        input.param, input.nidx, fidx, GradStats{l_gpair}, GradStats{r_gpair});
    best.Update(l_split, input.param);
    float parent_gain = CalcGain(input.param, input.parent_sum);  // FIXME: get it out
    float loss_chg = gain - parent_gain;
    float fvalue = input.feature_values[idx];
    if (forward) {
      bool replaced =
          best.Update(loss_chg, kRightDir, fvalue, fidx, GradientPair{l_gpair},
                      GradientPair{r_gpair}, is_cat, input.param);
      if (idx == 1) {
        printf("replaced: %d, be: %f, af: %f \n", int(replaced), best.loss_chg, loss_chg);
      }
    } else {
      best.Update(loss_chg, kLeftDir, fvalue, fidx, GradientPair{r_gpair},
                  GradientPair{l_gpair}, is_cat, input.param);
    }

    return best;
  }

  template <bool forward>
  XGBOOST_DEVICE ScanElem<GradientSumT> Scan(ScanElem<GradientSumT> const &l, ScanElem<GradientSumT> const &r) const {
    DeviceSplitCandidate l_split = l.candidate;

    if (l.idx < left.gradient_histogram.size()) {
      // Left node
      auto r_idx = r.idx;

      auto l_fidx = dh::SegmentId(left.feature_segments, l.idx);
      auto r_fidx = dh::SegmentId(left.feature_segments, r.idx);
      /* Segmented scan with 2 segments
       * *****|******
       * 0, 1 |  2, 3
       *   /|_|_/| /|
       * 0, 1 |  2, 5
       * *****|******
       */
      if (l_fidx != r_fidx) {
        // Segmented scan
        // if (forward) {
        //   printf("segment: l.idx: %lu, r.idx: %lu, %f\n", l.idx, r.idx, r.candidate.loss_chg);
        // }
        return r;
      }

      assert(!left.feature_set.empty());
      if ((left.feature_set.size() != left.feature_segments.size() - 1) &&
          !thrust::binary_search(thrust::seq, left.feature_set.cbegin(),
                                 left.feature_set.cend(), l_fidx)) {
        // column sampling
        return {r_idx, r.grad, DeviceSplitCandidate{}};
      }

      if (common::IsCat(left.feature_types, l_fidx)) {
        auto l_gpair = left.gradient_histogram[l.idx];
        auto r_gpair = left.parent_sum - l_gpair;
        auto best = DoIt<forward, true>(left, l.idx, l_gpair, r_gpair, l_split, l_fidx);
        return {r_idx, r_gpair, best};
      } else {
        auto l_gpair = l.grad;
        auto r_gpair = left.parent_sum - l_gpair;
        DeviceSplitCandidate best = DoIt<forward, false>(left, l.idx, l_gpair, r_gpair, l_split, l_fidx);
        // if (forward && best.IsValid() && l.idx != 0) {
        //   auto g = l_gpair + r.grad;
        //   printf("l_idx: %d, g: %f, h: %f, chg: %f\n", int(l.idx), l_gpair.GetGrad(), l_gpair.GetHess(), best.loss_chg);
        // }
        return {r_idx, l_gpair + r.grad, best};
      }
    } else {
      // Right node
      if (left.gradient_histogram.size() != right.gradient_histogram.size()) {
        printf("left.gradient_histogram.size(): %lu, "
               "right.gradient_histogram.size(): %lu, idx: %lu, forward: %d\n",
               left.gradient_histogram.size(), right.gradient_histogram.size(),
               l.idx, int(forward));
      }
      assert(left.gradient_histogram.size() == right.gradient_histogram.size());
      auto l_idx = l.idx - left.gradient_histogram.size();
      auto r_idx = r.idx - left.gradient_histogram.size();

      auto l_fidx = dh::SegmentId(right.feature_segments, l_idx);
      auto r_fidx = dh::SegmentId(right.feature_segments, r_idx);
      if (l_fidx != r_fidx) {
        // Segmented scan
        return {r.idx, r.grad, r.candidate};
      }

      assert(!right.feature_segments.empty());
      if ((right.feature_set.size() != right.feature_segments.size()) &&
          !thrust::binary_search(thrust::seq, right.feature_set.cbegin(),
                                 right.feature_set.cend(), l_fidx)) {
        // column sampling
        return {r_idx, r.grad, DeviceSplitCandidate{}};
      }

      if (common::IsCat(right.feature_types, l_fidx)) {
        auto l_gpair = right.gradient_histogram[l_idx];
        auto r_gpair = right.parent_sum - l_gpair;
        auto best = DoIt<forward, true>(right, l_idx, l_gpair, r_gpair, l_split, l_fidx);
        return {r_idx, r_gpair, best};
      } else {
        auto l_gpair = l.grad;
        auto r_gpair = right.parent_sum - l_gpair;
        auto best = DoIt<forward, false>(right, l_idx, l_gpair, r_gpair, l_split, l_fidx);
        return {r_idx, l.grad + r.grad, best};
      }
    }
  }

  using Ty = thrust::tuple<ItemTy, ItemTy>;

  XGBOOST_DEVICE Ty operator()(Ty const &l, Ty const &r) const {
    auto fw = Scan<true>(thrust::get<0>(l), thrust::get<0>(r));
    auto bw = Scan<false>(thrust::get<1>(l), thrust::get<1>(r));
    return thrust::make_tuple(fw, bw);
  }
};

template <typename GradientSumT, typename Tu = thrust::tuple<ScanElem<GradientSumT>, ScanElem<GradientSumT>>>
class DiscardOverload : public thrust::discard_iterator<Tu> {
 public:
  using value_type = Tu;  // NOLINT
};

template <typename GradientSumT>
struct WriteScan {
  using Tuple = typename DiscardOverload<GradientSumT>::value_type;
  using ScanElemTy = ScanElem<GradientSumT>;
  EvaluateSplitInputs<GradientSumT> left;
  EvaluateSplitInputs<GradientSumT> right;
  common::Span<ScanElem<GradientSumT>> d_out_scan;
  size_t n_features;

  template <bool forward>
  XGBOOST_DEVICE void DoIt(ScanElemTy const& candidate) {
    size_t offset = 0;
    size_t beg_idx = 0;
    size_t end_idx = 0;

    auto fidx = candidate.candidate.findex;
    auto idx = candidate.idx;

    if (idx < left.gradient_histogram.size()) {
      // left node
      beg_idx = left.feature_segments[fidx];
      auto f_size = left.feature_segments[fidx + 1] - beg_idx;
      f_size = f_size == 0 ? 0 : f_size - 1;
      end_idx = beg_idx + f_size;
    } else {
      // right node
      beg_idx = right.feature_segments[fidx];
      auto f_size = right.feature_segments[fidx + 1] - beg_idx;
      f_size = f_size == 0 ? 0 : f_size - 1;
      end_idx = beg_idx + f_size;
      offset = n_features * 2;
    }
    // printf("idx: %lu, fidx: %d, beg: %lu, end: %lu, chg: %f, g: %f, f: %d \n",
    //        idx, int(fidx), beg_idx, end_idx, candidate.candidate.loss_chg,
    //        candidate.grad.GetGrad(), int(forward));
    if (forward) {
      if (end_idx == idx) {
        d_out_scan[offset + fidx] = candidate;
      }
    } else {
      if (beg_idx == idx) {
        d_out_scan[offset + n_features + fidx] = candidate;
      }
    }
  }

  XGBOOST_DEVICE Tuple operator()(Tuple const &tu) {
    ScanElem<GradientSumT> const &fw = thrust::get<0>(tu);
    ScanElem<GradientSumT> const &bw = thrust::get<1>(tu);
    if (fw.candidate.findex != -1) {
      DoIt<true>(fw);
    }
    if (bw.candidate.findex != -1) {
      DoIt<false>(bw);
    }
    return {};  // discard
  }
};

template <typename GradientSumT, bool forward>
struct ScanValueOp {
  EvaluateSplitInputs<GradientSumT> left;
  EvaluateSplitInputs<GradientSumT> right;
  TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator;

  XGBOOST_DEVICE ScanElem<GradientSumT> operator()(size_t idx) {
    ScanElem<GradientSumT> ret;
    ret.idx = idx;
    float fvalue;
    size_t fidx;
    bool is_cat;
    float loss_chg;
    if (idx < left.gradient_histogram.size()) {
      // left node
      ret.grad = left.gradient_histogram[idx];
      fvalue = left.feature_values[idx];
      fidx = dh::SegmentId(left.feature_segments, idx);
      is_cat = common::IsCat(left.feature_types, fidx);

      float parent_gain =
          CalcGain(left.param, left.parent_sum); // FIXME: get it out
      float gain = evaluator.CalcSplitGain(left.param, left.nidx, fidx,
                                           GradStats{ret.grad},
                                           GradStats{left.parent_sum - ret.grad});
      loss_chg = gain - parent_gain;
    } else {
      // right node
      idx -= left.gradient_histogram.size();
      ret.grad = right.gradient_histogram[idx];
      fvalue = right.feature_values[idx];
      fidx = dh::SegmentId(right.feature_segments, idx);
      is_cat = common::IsCat(right.feature_types, fidx);

      float parent_gain =
          CalcGain(right.param, right.parent_sum); // FIXME: get it out
      float gain = evaluator.CalcSplitGain(right.param, right.nidx, fidx,
                                           GradStats{ret.grad},
                                           GradStats{left.parent_sum - ret.grad});
      loss_chg = gain - parent_gain;
    };
    if (forward) {
      ret.candidate.Update(
          loss_chg, kRightDir, fvalue, fidx, GradientPair{ret.grad},
          GradientPair{left.parent_sum - ret.grad}, is_cat, left.param);
    } else {
      ret.candidate.Update(loss_chg, kLeftDir, fvalue, fidx,
                           GradientPair{left.parent_sum - ret.grad},
                           GradientPair{ret.grad}, is_cat, left.param);
    }
    return ret;
  }
};
}  // anonymous namespace

template <typename GradientSumT>
void EvaluateSplits(common::Span<DeviceSplitCandidate> out_splits,
                    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
                    EvaluateSplitInputs<GradientSumT> left,
                    EvaluateSplitInputs<GradientSumT> right) {
  /**
   * Validate inputs
   */
  CHECK(left.gradient_histogram.size() == right.gradient_histogram.size() ||
        right.gradient_histogram.empty());
  CHECK(left.feature_segments.size() == right.feature_segments.size() ||
        right.feature_segments.empty());
  if (left.feature_segments.empty()) {
    CHECK(left.gradient_histogram.empty());
  }
  if (right.feature_segments.empty()) {
    CHECK(right.gradient_histogram.empty());
  }

  std::cout << "left: " << left.gradient_histogram.size()
            << ", right: " << right.gradient_histogram.size() << std::endl;

  auto l_n_features = left.feature_segments.empty() ? 0 : left.feature_segments.size() - 1;
  auto r_n_features = right.feature_segments.empty() ? 0 : right.feature_segments.size() - 1;
  CHECK(r_n_features == 0 || l_n_features == r_n_features);
  auto n_features = l_n_features + r_n_features;

  /**
   * Handle trivial input
   */
  if (n_features == 0) {
    dh::LaunchN(out_splits.size(), [=]XGBOOST_DEVICE(size_t idx) {
      out_splits[idx] = DeviceSplitCandidate{};
    });
    return;
  }

  std::cout << "n_features:" << n_features << std::endl;

  size_t size = left.gradient_histogram.size() + right.gradient_histogram.size();

  auto for_counting = thrust::make_counting_iterator(0ul);
  auto rev_counting =
      thrust::make_reverse_iterator(thrust::make_counting_iterator(size));
  auto for_value_iter = dh::MakeTransformIterator<ScanElem<GradientSumT>>(
      for_counting, ScanValueOp<GradientSumT, true>{left, right, evaluator});
  auto rev_value_iter = dh::MakeTransformIterator<ScanElem<GradientSumT>>(
      rev_counting, ScanValueOp<GradientSumT, false>{left, right, evaluator});

  auto value_iter = thrust::make_zip_iterator(thrust::make_tuple(for_value_iter, rev_value_iter));
  using FBTuple = thrust::tuple<ScanElem<GradientSumT>, ScanElem<GradientSumT>>;
  dh::device_vector<ScanElem<GradientSumT>> out_scan(n_features * 2); // x2 due to forward and backward
  auto d_out_scan = dh::ToSpan(out_scan);

  using Discard = DiscardOverload<GradientSumT>;
  auto out_it = thrust::make_transform_output_iterator(
      Discard(), WriteScan<GradientSumT>{left, right, d_out_scan, l_n_features});

  size_t temp_bytes = 0;
  hipcub::DeviceScan::InclusiveScan(nullptr, temp_bytes, value_iter, out_it,
                                 ScanOp<GradientSumT>{left, right, evaluator},
                                 size);
  dh::TemporaryArray<int8_t> temp(temp_bytes);
  hipcub::DeviceScan::InclusiveScan(
      temp.data().get(), temp_bytes, value_iter, out_it,
      ScanOp<GradientSumT>{left, right, evaluator}, size);

  // {
  //   // debug
  //   std::cout << "beg debug" << std::endl;
  //   dh::device_vector<FBTuple> out_scan(size);
  //   size_t temp_bytes = 0;
  //   hipcub::DeviceScan::InclusiveScan(nullptr, temp_bytes, value_iter, out_scan.begin(),
  //                                  ScanOp<GradientSumT>{left, right, evaluator},
  //                                  size);
  //   dh::TemporaryArray<int8_t> temp(temp_bytes);
  //   hipcub::DeviceScan::InclusiveScan(
  //       temp.data().get(), temp_bytes, value_iter, out_scan.begin(),
  //       ScanOp<GradientSumT>{left, right, evaluator}, size);
  //   dh::DebugSyncDevice();
  //   for (size_t i = 0; i < out_scan.size(); ++i) {
  //     auto fw = thrust::get<0>(FBTuple(out_scan[i]));
  //     auto bw = thrust::get<1>(FBTuple(out_scan[i]));
  //     std::cout << "i: " << i << ", grad: " << fw.grad << "\n"
  //               << fw.candidate << bw.candidate << std::endl;
  //   }
  //   std::cout << "end debug" << std::endl;
  // }

  dh::DebugSyncDevice();
  for (size_t i = 0; i < out_scan.size(); ++i) {
    auto candidate = ScanElem<GradientSumT>(out_scan[i]);
    std::cout << "i: " << i << ", grad: " << candidate.grad << "\n"
              << candidate.candidate << std::endl;
  }

  auto reduce_key = dh::MakeTransformIterator<int>(
      thrust::make_counting_iterator(0ul),
      [=] XGBOOST_DEVICE(bst_feature_t fidx) -> int {
        if (fidx < l_n_features * 2) {
          return 0;  // left node
        } else {
          return 1;  // right node
        }
      });
  auto reduce_val = dh::MakeTransformIterator<DeviceSplitCandidate>(
      thrust::make_counting_iterator(0),
      [d_out_scan] __device__(size_t idx) {
        // No need to distinguish left and right node as we are just extracting values.
        ScanElem<GradientSumT> candidate = d_out_scan[idx];
        return candidate.candidate;
      });
  thrust::reduce_by_key(
      thrust::device, reduce_key, reduce_key + out_scan.size(),
      reduce_val, thrust::make_discard_iterator(), out_splits.data(),
      thrust::equal_to<int>{},
      [=] XGBOOST_DEVICE(DeviceSplitCandidate l, DeviceSplitCandidate r) {
        l.Update(r, left.param);
        return l;
      });
}

template <typename GradientSumT>
void EvaluateSingleSplit(common::Span<DeviceSplitCandidate> out_split,
                         TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
                         EvaluateSplitInputs<GradientSumT> input) {
  EvaluateSplits(out_split, evaluator, input, {});
}

template void EvaluateSplits<GradientPair>(
    common::Span<DeviceSplitCandidate> out_splits,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    EvaluateSplitInputs<GradientPair> left,
    EvaluateSplitInputs<GradientPair> right);
template void EvaluateSplits<GradientPairPrecise>(
    common::Span<DeviceSplitCandidate> out_splits,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    EvaluateSplitInputs<GradientPairPrecise> left,
    EvaluateSplitInputs<GradientPairPrecise> right);
template void EvaluateSingleSplit<GradientPair>(
    common::Span<DeviceSplitCandidate> out_split,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    EvaluateSplitInputs<GradientPair> input);
template void EvaluateSingleSplit<GradientPairPrecise>(
    common::Span<DeviceSplitCandidate> out_split,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    EvaluateSplitInputs<GradientPairPrecise> input);
}  // namespace tree
}  // namespace xgboost
